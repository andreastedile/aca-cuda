#include "hip/hip_runtime.h"
#include "utils.h"

#include <cmath>
#include <cstdio>

/**
* Assumes that n is a power of four
* @return base 4 integer logarithm of n
*/
CUDA_HOSTDEV int log4(int n) {
    // same as log(n) / log(4)
    return static_cast<int>(std::log2(n) / 2);
}

CUDA_HOSTDEV int pow4(int n) {
    // same as pow(4, n)
    return int(std::pow(4, n));
}

bool is_power_of_four(int n) {
    return n == std::pow(4, log4(n));
}

void CHECK(hipError_t error) {
    if (error != hipSuccess) {
        fprintf(stderr, "Got error %s at %s:%d\n", hipGetErrorString(error), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}
